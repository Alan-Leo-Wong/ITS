#include "hip/hip_runtime.h"
#include "MCDefine.h"
#include "MarchingCubes.h"
#include "..\CUDACompute.h"
#include "..\..\BSpline.hpp"
#include <>

/**
 * @brief ׼�����ڼ��� sdf �ľ���
 *
 * @param nVoxelElems   ����ÿ�� stream �� voxel ����Ŀ nVoxelElems * 8��ÿ�� stream �� voxel �Ķ�����
 * @param nAllNodes     ���� nAllNodes * 8�����нڵ�Ķ�����
 * @param d_voxelOffset ��ǰ���� voxel ��������� voxel ��ƫ����
 * @param d_res         �ֱ���
 * @param d_lambda      B����ϵ��
 * @param d_origin      MC�㷨��ִ�еĳ�ʼ����ԭ������
 * @param d_voxelSize   ÿ�� voxel �Ĵ�С
 * @param d_nodeCorners ÿ���˲����ڵ�Ķ���
 * @param d_nodeWidth   ÿ���˲����ڵ�Ŀ��
 * @param d_voxelMatrix output matrix
 */
__global__ void MCKernel::prepareMatrixKernel(const uint nVoxelElems,
	const uint nAllNodes,
	const uint* d_voxelOffset,
	const uint3* d_res,
	double* d_lambda,
	double3* d_origin,
	double3* d_voxelSize,
	V3d* d_nodeCorners,
	V3d* d_nodeWidth,
	double* d_voxelMatrix)
{
	uint tx = blockIdx.x * blockDim.x + threadIdx.x;
	uint ty = blockIdx.y * blockDim.y + threadIdx.y;

	if (tx < nAllNodes && ty < nVoxelElems)
	{
		const uint voxel = ty + (*d_voxelOffset);

		uint3 voxelShift = getVoxelShift(voxel, *d_res);
		double3 origin = *d_origin;
		double3 voxelSize = *d_voxelSize;
		double3 voxelPos; // voxel ���º������

		voxelPos.x = origin.x + voxelShift.x * voxelSize.x;
		voxelPos.y = origin.y + voxelShift.y * voxelSize.y;
		voxelPos.z = origin.z + voxelShift.z * voxelSize.z;

		// corners of current voxel
		double3 corners[8];
		corners[0] = voxelPos;
		corners[1] = voxelPos + make_double3(0, voxelSize.y, 0);
		corners[2] = voxelPos + make_double3(voxelSize.x, voxelSize.y, 0);
		corners[3] = voxelPos + make_double3(voxelSize.x, 0, 0);
		corners[4] = voxelPos + make_double3(0, 0, voxelSize.z);
		corners[5] = voxelPos + make_double3(0, voxelSize.y, voxelSize.z);
		corners[6] = voxelPos + make_double3(voxelSize.x, voxelSize.y, voxelSize.z);
		corners[7] = voxelPos + make_double3(voxelSize.x, 0, voxelSize.z);

		V3d width = d_nodeWidth[tx];

		for (int k = 0; k < 8; ++k)
		{
			const int nodeCornerIdx = tx * 8 + k;
			double3 corner = corners[nodeCornerIdx];
			const int idx = (voxel + k) * nAllNodes * 8 + nodeCornerIdx;

			d_voxelMatrix[idx] = d_lambda[nodeCornerIdx] * BaseFunction4Point(d_nodeCorners[nodeCornerIdx], width, V3d(corner.x, corner.y, corner.z));
		}
	}
}

void MC::launch_prepareMatrixKernel(const uint& nVoxelElems, const uint& voxelOffset, const hipStream_t& stream, double* d_voxelMatrix)
{
	uint* d_voxelOffset = nullptr;
	CUDA_CHECK(hipMalloc((void**)&d_voxelOffset, sizeof(uint)));
	CUDA_CHECK(hipMemcpyAsync(d_voxelOffset, &voxelOffset, sizeof(uint), hipMemcpyHostToDevice, stream));

	// �������(nVoxelElemCorners * 8 * nAllNodes * 8)
	const uint nVoxelElemCorners = nVoxelElems * 8;
	const uint nAllNodeCorners = nAllNodes * 8;
	uint voxelMatrixSize = nVoxelElemCorners * nAllNodeCorners;
	CUDA_CHECK(hipMalloc((void**)&d_voxelMatrix, sizeof(double) * voxelMatrixSize));

	dim3 nThreads(P_NTHREADS_X, P_NTHREADS_Y, 1);
	assert(P_NTHREADS_X * P_NTHREADS_Y <= 1024, "P_NTHREADS_X * P_NTHREADS_Y is larger than 1024!\n");
	dim3 nBlocks((nAllNodes + nThreads.x - 1) / nThreads.x, (nVoxelElems + nThreads.y - 1) / nThreads.y, 1);

	MCKernel::prepareMatrixKernel << <nBlocks, nThreads >> > (nVoxelElems, nAllNodes,
		d_voxelOffset, d_res, d_lambda, d_gridOrigin,
		d_voxelSize, d_nodeCorners, d_nodeWidth, d_voxelMatrix);

	CUDA_CHECK(hipFree(d_voxelOffset));
}

void MC::launch_computSDFKernel(const uint& nVoxels)
{
	hipStream_t streams[MAX_NUM_STREAMS];
	for (int i = 0; i < MAX_NUM_STREAMS; ++i)
		CUDA_CHECK(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));

	for (int i = 0; i < MAX_NUM_STREAMS; ++i)
	{
		// nVoxelElems: the number of voxels for each stream
		uint nVoxelElems = (nVoxels + MAX_NUM_STREAMS - 1) / MAX_NUM_STREAMS;
		uint voxelOffset = i * nVoxelElems;
		nVoxelElems = voxelOffset + nVoxelElems > nVoxels ? nVoxels - voxelOffset : nVoxelElems;

		double* d_voxelMatrix = nullptr;

		launch_prepareMatrixKernel(nVoxelElems, voxelOffset, streams[i], d_voxelMatrix);

		launch_BLASRowSumReduce(streams[i], nVoxelElems * 8, nAllNodes * 8, d_voxelMatrix, d_voxelSDF + voxelOffset * 8);

		CUDA_CHECK(hipFree(d_voxelMatrix));
	}

	for (int i = 0; i < MAX_NUM_STREAMS; i++)
		hipStreamSynchronize(streams[i]);
	for (int i = 0; i < MAX_NUM_STREAMS; ++i)
		CUDA_CHECK(hipStreamDestroy(streams[i]));
}